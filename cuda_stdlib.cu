#include "hip/hip_runtime.h"
#define QUDA_GATE __device__
#define CUSTOM_HADAMARD
#define FOR_EACH_STATE(qreg, i) \
  for (i = blockIdx.x * blockDim.x + threadIdx.x; i < qreg->num_states; \
      i += blockDim.x * gridDim.x)
#include "quantum_reg.h"
#include "complex.c"
#include "quantum_gates.c"

// Magic for hadamard gate
__device__ int quda_quantum_hadamard_gate(int target, quantum_reg* qreg) {
	// If needed, enlarge qreg to make room for state splits resulting from this gate
  int states = qreg->num_states;
  if (2 * states > qreg->size) {
    *((volatile int *)0) = 0;
    return -1;
  }

	uint64_t mask = 1 << target;
	int i;
	FOR_EACH_STATE(qreg, i) {
		// Flipped state must be created
		qreg->states[qreg->num_states+i].state = qreg->states[i].state ^ mask;
		// For this state, must just modify amplitude
		qreg->states[i].amplitude = quda_complex_rmul(qreg->states[i].amplitude,
				ONE_OVER_SQRT_2);
		// Copy amplitude to created state
		qreg->states[qreg->num_states+i].amplitude = qreg->states[i].amplitude;

		if(qreg->states[i].state & mask) {
			qreg->states[i].amplitude = quda_complex_neg(qreg->states[i].amplitude);
		}
	}

  __syncthreads();
  if (blockIdx.x == 0 && threadIdx.x == 0)
  	qreg->num_states = 2*states;
  __syncthreads();

	// TODO: Ideally, make this call optional or conditional
	quda_quantum_reg_coalesce(qreg);

	return 0;
}
__device__ void quda_quantum_reg_coalesce(quantum_reg *qreg) {
  __syncthreads();
}

__global__ void cuda_quantum_fourier_kernel(quantum_reg *qreg) {
	int q = qreg->qubits-1;
	int i,j;
	for(i=q;i>=0;i--) {
		for(j=q;j>i;j--) {
			#ifdef QUDA_STDLIB_DEBUG
			printf("Performing c-R_%d (PI/%lu) on (%d,%d)\n",j-i+1,(uint64_t)1 << (j-i),j,i); // DEBUG
			#endif
			quda_quantum_controlled_rotate_k_gate(j,i,qreg,j-i+1);
		}
		#ifdef QUDA_STDLIB_DEBUG
		printf("Performing hadamard(bit %d)\n",i); // DEBUG
		#endif
		quda_quantum_hadamard_gate(i,qreg);
	}

	// TODO: Consider using SWAP gate here instead
	for(i=0;i<qreg->qubits/2;i++) {
		quda_quantum_controlled_not_gate(i,q-i,qreg);
		quda_quantum_controlled_not_gate(q-i,i,qreg);
		quda_quantum_controlled_not_gate(i,q-i,qreg);
	}
}

#include <stdio.h>
#define SANITY_CHECK(err) \
  do { \
    hipError_t err__ = err; \
    if (err__ != hipSuccess) { \
      fprintf(stderr, "Error at line %d: %s\n", __LINE__, \
          hipGetErrorString(err__)); \
      exit(-1); \
    } \
  } while (0)
extern "C" void quda_cu_quantum_fourier_transform(quantum_reg* qreg) {
  quantum_state_t *qstates_device;
  quantum_reg qreg_host = *qreg, *qreg_device;
  hipError_t err;

  // Copy over the states to the device
  err = hipMalloc(&qstates_device, 1LL << qreg->size);
  SANITY_CHECK(err);
  err = hipMalloc(&qreg_device, sizeof(quantum_reg));
  SANITY_CHECK(err);
  err = hipMemcpy(qstates_device, qreg->states,
    qreg->num_states * sizeof(quantum_state_t), hipMemcpyHostToDevice);
  SANITY_CHECK(err);
  qreg_host.states = qstates_device;

  // Copy over the device pointer for qreg
  err = hipMemcpy(qreg_device, &qreg_host, sizeof(quantum_reg),
    hipMemcpyHostToDevice);
  SANITY_CHECK(err);

  // Invoke the kernel
  dim3 localSize(128, 1, 1);
  dim3 globalSize(1, 1, 1);
  cuda_quantum_fourier_kernel<<<globalSize, localSize>>>(qreg_device);
  SANITY_CHECK(hipGetLastError());
  // Free the memory locally (we'll replace it slightly later)
  free(qreg->states);

  // Copy back the device pointer
  err = hipMemcpy(qreg, qreg_device, sizeof(quantum_reg),
    hipMemcpyDeviceToHost);
  SANITY_CHECK(err);
  // ... and the states
  qreg->states = (quantum_state_t*)malloc(sizeof(quantum_state_t) * qreg->num_states);
  err = hipMemcpy(qreg->states, qstates_device,
    sizeof(quantum_state_t) * qreg->num_states, hipMemcpyDeviceToHost);
  SANITY_CHECK(err);
}
