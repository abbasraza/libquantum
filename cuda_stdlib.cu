#include "hip/hip_runtime.h"
#define QUDA_GATE __device__
#include "quantum_reg.h"
#include "complex.c"
#include "quantum_gates.c"

// Magic for hadamard gate
__device__ int quda_quantum_reg_enlarge(quantum_reg *qreg, int amount) {
  if (qreg->size < amount) {
    *((volatile int *)0) = 0;
    return -1;
  }
  return 0;
}

__device__ void quda_quantum_reg_coalesce(quantum_reg *qreg) {
}

__global__ void cuda_quantum_fourier_kernel(quantum_reg *qreg) {
	int q = qreg->qubits-1;
	int i,j;
	for(i=q;i>=0;i--) {
		for(j=q;j>i;j--) {
			#ifdef QUDA_STDLIB_DEBUG
			printf("Performing c-R_%d (PI/%lu) on (%d,%d)\n",j-i+1,(uint64_t)1 << (j-i),j,i); // DEBUG
			#endif
			quda_quantum_controlled_rotate_k_gate(j,i,qreg,j-i+1);
		}
		#ifdef QUDA_STDLIB_DEBUG
		printf("Performing hadamard(bit %d)\n",i); // DEBUG
		#endif
		quda_quantum_hadamard_gate(i,qreg);
	}

	// TODO: Consider using SWAP gate here instead
	for(i=0;i<qreg->qubits/2;i++) {
		quda_quantum_controlled_not_gate(i,q-i,qreg);
		quda_quantum_controlled_not_gate(q-i,i,qreg);
		quda_quantum_controlled_not_gate(i,q-i,qreg);
	}
}

#include <stdio.h>
#define SANITY_CHECK(err) \
  do { \
    hipError_t err__ = err; \
    if (err__ != hipSuccess) { \
      fprintf(stderr, "Error at line %d: %s\n", __LINE__, \
          hipGetErrorString(err__)); \
      exit(-1); \
    } \
  } while (0)
extern "C" void quda_cu_quantum_fourier_transform(quantum_reg* qreg) {
  quantum_state_t *qstates_device;
  quantum_reg qreg_host = *qreg, *qreg_device;
  hipError_t err;

  // Copy over the states to the device
  err = hipMalloc(&qstates_device, 1LL << qreg->size);
  SANITY_CHECK(err);
  err = hipMalloc(&qreg_device, sizeof(quantum_reg));
  SANITY_CHECK(err);
  err = hipMemcpy(qstates_device, qreg->states,
    qreg->num_states * sizeof(quantum_state_t), hipMemcpyHostToDevice);
  SANITY_CHECK(err);
  qreg_host.states = qstates_device;

  // Copy over the device pointer for qreg
  err = hipMemcpy(qreg_device, &qreg_host, sizeof(quantum_reg),
    hipMemcpyHostToDevice);
  SANITY_CHECK(err);

  // Invoke the kernel
  dim3 localSize(1, 1, 1);
  dim3 globalSize(1, 1, 1);
  cuda_quantum_fourier_kernel<<<globalSize, localSize>>>(qreg_device);
  SANITY_CHECK(hipGetLastError());
  // Free the memory locally (we'll replace it slightly later)
  free(qreg->states);

  // Copy back the device pointer
  err = hipMemcpy(qreg, qreg_device, sizeof(quantum_reg),
    hipMemcpyDeviceToHost);
  SANITY_CHECK(err);
  // ... and the states
  qreg->states = (quantum_state_t*)malloc(sizeof(quantum_state_t) * qreg->num_states);
  err = hipMemcpy(qreg->states, qstates_device,
    sizeof(quantum_state_t) * qreg->num_states, hipMemcpyDeviceToHost);
  SANITY_CHECK(err);
}
