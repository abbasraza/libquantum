#include "hip/hip_runtime.h"
#define QUDA_GATE __device__
#define CUSTOM_HADAMARD
#define FOR_EACH_STATE(qreg, i) \
  for (i = blockIdx.x * blockDim.x + threadIdx.x; i < qreg->num_states; \
      i += blockDim.x * gridDim.x)
#define STATE(qreg, i) qreg->states[i]
#define AMPLITUDE(qreg, i) qreg->amplitudes[i]

#include "quantum_reg.h"
#include "complex.c"
typedef struct {
  int num_states;
  int size;
  int qubits;
  uint64_t *states;
  complex_t *amplitudes;
} cuda_quantum_reg;

#define quantum_reg cuda_quantum_reg
#include "quantum_gates.c"
#undef quantum_reg

// Magic for hadamard gate
__device__ int quda_quantum_hadamard_gate(int target, cuda_quantum_reg* qreg) {
	// If needed, enlarge qreg to make room for state splits resulting from this gate
  int states = qreg->num_states;
  if (2 * states > qreg->size) {
    *((volatile int *)0) = 0;
    return -1;
  }

	uint64_t mask = 1 << target;
	int i;
	FOR_EACH_STATE(qreg, i) {
		// Flipped state must be created
		STATE(qreg, qreg->num_states+i) = STATE(qreg, i) ^ mask;
		// For this state, must just modify amplitude
		AMPLITUDE(qreg, i) = quda_complex_rmul(AMPLITUDE(qreg, i),
				ONE_OVER_SQRT_2);
		// Copy amplitude to created state
		AMPLITUDE(qreg, qreg->num_states+i) = AMPLITUDE(qreg, i);

		if(STATE(qreg, i) & mask) {
			AMPLITUDE(qreg, i) = quda_complex_neg(AMPLITUDE(qreg, i));
		}
	}


  __syncthreads();
  if (blockIdx.x == 0 && threadIdx.x == 0)
  	qreg->num_states = 2*states;
  __syncthreads();

	return 0;
}

#include <stdio.h>
__global__ void cuda_quantum_fourier_kernel(cuda_quantum_reg *qreg) {
	int q = qreg->qubits-1;
	int i,j;
	for(i=q;i>=0;i--) {
		for(j=q;j>i;j--) {
			#ifdef QUDA_STDLIB_DEBUG
			printf("Performing c-R_%d (PI/%lu) on (%d,%d)\n",j-i+1,(uint64_t)1 << (j-i),j,i); // DEBUG
			#endif
			quda_quantum_controlled_rotate_k_gate(j,i,qreg,j-i+1);
		}
		#ifdef QUDA_STDLIB_DEBUG
		printf("Performing hadamard(bit %d)\n",i); // DEBUG
		#endif
		quda_quantum_hadamard_gate(i,qreg);
	}

	// TODO: Consider using SWAP gate here instead
	for(i=0;i<qreg->qubits/2;i++) {
		quda_quantum_controlled_not_gate(i,q-i,qreg);
		quda_quantum_controlled_not_gate(q-i,i,qreg);
		quda_quantum_controlled_not_gate(i,q-i,qreg);
	}
}

#define SANITY_CHECK(err) \
  do { \
    hipError_t err__ = err; \
    if (err__ != hipSuccess) { \
      fprintf(stderr, "Error at line %d: %s\n", __LINE__, \
          hipGetErrorString(err__)); \
      exit(-1); \
    } \
  } while (0)
extern "C" void quda_cu_quantum_fourier_transform(quantum_reg* qreg) {
  uint64_t *states_device;
  complex_t *amplitudes_device;
  cuda_quantum_reg qreg_host, *qreg_device;
  hipError_t err;

  // Copy over the states to the device
  qreg_host.size = 1 << qreg->qubits;
  qreg_host.num_states = qreg->num_states;
  qreg_host.qubits = qreg->qubits;
  err = hipMalloc(&states_device, sizeof(uint64_t) * qreg_host.size);
  SANITY_CHECK(err);
  err = hipMalloc(&amplitudes_device, sizeof(complex_t) * qreg_host.size);
  SANITY_CHECK(err);
  err = hipMalloc(&qreg_device, sizeof(cuda_quantum_reg));
  SANITY_CHECK(err);

  // ... async!
  hipStream_t stream;
  err = hipStreamCreate(&stream);
  SANITY_CHECK(err);
  err = hipMemcpy2DAsync(states_device, sizeof(uint64_t),
    &qreg->states[0].state, sizeof(uint64_t) + sizeof(complex_t),
    sizeof(uint64_t), qreg->num_states, hipMemcpyHostToDevice, stream);
  SANITY_CHECK(err);
  err = hipMemcpy2DAsync(amplitudes_device, sizeof(complex_t),
    &qreg->states[0].amplitude, sizeof(uint64_t) + sizeof(complex_t),
    sizeof(complex_t), qreg->num_states, hipMemcpyHostToDevice, stream);
  SANITY_CHECK(err);
  qreg_host.states = states_device;
  qreg_host.amplitudes = amplitudes_device;

  // Copy over the device pointer for qreg
  err = hipMemcpyAsync(qreg_device, &qreg_host, sizeof(cuda_quantum_reg),
    hipMemcpyHostToDevice, stream);
  SANITY_CHECK(err);

  // Note for when we can stop waiting
  hipEvent_t gate;
  err = hipEventCreate(&gate);
  SANITY_CHECK(err);

  // Invoke the kernel
  dim3 localSize(128, 1, 1);
  dim3 globalSize(1, 1, 1);
  cuda_quantum_fourier_kernel<<<globalSize, localSize, 0, stream>>>(qreg_device);
  SANITY_CHECK(hipGetLastError());
  // Free the memory locally (we'll replace it slightly later)
  err = hipStreamWaitEvent(stream, gate, 0);
  SANITY_CHECK(err);
  err = hipEventDestroy(gate);
  SANITY_CHECK(err);
  free(qreg->states);

  // Copy back the device pointer
  err = hipMemcpyAsync(&qreg_host, qreg_device, sizeof(cuda_quantum_reg),
    hipMemcpyDeviceToHost, stream);
  SANITY_CHECK(err);
  qreg->size = qreg->num_states = qreg_host.num_states;
  // ... and the states
  qreg->states = (quantum_state_t*)malloc(sizeof(quantum_state_t) * qreg->num_states);
  err = hipMemcpy2DAsync(&qreg->states[0].state,
    sizeof(uint64_t) + sizeof(complex_t),
    states_device, sizeof(uint64_t),
    sizeof(uint64_t), qreg->num_states, hipMemcpyDeviceToHost, stream);
  SANITY_CHECK(err);
  err = hipMemcpy2DAsync(&qreg->states[0].amplitude,
    sizeof(uint64_t) + sizeof(complex_t),
    amplitudes_device, sizeof(complex_t),
    sizeof(complex_t), qreg->num_states, hipMemcpyDeviceToHost, stream);
  SANITY_CHECK(err);

  err = hipStreamSynchronize(stream);
  SANITY_CHECK(err);
  err = hipStreamDestroy(stream);
  SANITY_CHECK(err);
}
